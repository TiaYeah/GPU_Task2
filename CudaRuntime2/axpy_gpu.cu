#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <vector>

#include <stdio.h>
#include <omp.h>
#include "axpy_gpu.h"

__global__ void saxpy_gpuKernel(int n, float a, float* x, int incx, float* y, int incy)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i * incy] = y[i * incy] + a * x[i * incx];
    }
}

__global__ void daxpy_gpuKernel(int n, double a, double* x, int incx, double* y, int incy)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i * incy] = y[i * incy] + a * x[i * incx];
    }
}

void saxpy_gpu(int n, float a, float* x, int incx, float* y, int incy, int block_size) {
    float* x_gpu;
    float* y_gpu;
    hipError_t cudaStatus;
    
    
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    
    // Allocate GPU buffers for three vectors (two input, one output)    .
    
    int x_gpu_size = 1 + (n - 1) * incx;
    int y_gpu_size = 1 + (n - 1) * incy;

    cudaStatus = hipMalloc((void**)&x_gpu, x_gpu_size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&y_gpu, y_gpu_size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(x_gpu, x, x_gpu_size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    cudaStatus = hipMemcpy(y_gpu, y, y_gpu_size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    // Launch a kernel on the GPU with one thread for each element.
    //const int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;
    double begin, end;

    begin = omp_get_wtime();
    saxpy_gpuKernel <<<num_blocks, block_size >>> (n, a, x_gpu, incx, y_gpu, incy);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "saxpy_gpu launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    end = omp_get_wtime();

    printf("GPU time with block size %d: %f\n", block_size, end - begin);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching saxpy_gpu!\n", cudaStatus);
        goto Error;
    }
    
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(y, y_gpu, y_gpu_size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
Error:
    hipFree(x_gpu);
    hipFree(y_gpu);
        
}

void daxpy_gpu(int n, double a, double* x, int incx, double* y, int incy, int block_size) {
    double* x_gpu;
    double* y_gpu;
    hipError_t cudaStatus;


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .

    int x_gpu_size = 1 + (n - 1) * incx;
    int y_gpu_size = 1 + (n - 1) * incy;

    cudaStatus = hipMalloc((void**)&x_gpu, x_gpu_size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&y_gpu, y_gpu_size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(x_gpu, x, x_gpu_size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(y_gpu, y, y_gpu_size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    int num_blocks = (n + block_size - 1) / block_size;
    double begin, end;

    begin = omp_get_wtime();
    daxpy_gpuKernel << <num_blocks, block_size >> > (n, a, x_gpu, incx, y_gpu, incy);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "daxpy_gpu launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    end = omp_get_wtime();

    printf("GPU time with block size %d: %f\n", block_size, end - begin);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching daxpy_gpu!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(y, y_gpu, y_gpu_size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(x_gpu);
    hipFree(y_gpu);

}
